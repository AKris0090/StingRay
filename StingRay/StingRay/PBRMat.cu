#include "PBRMat.cuh"

__device__ V3 PBRMaterial::random_direction(hiprandState* localDevState) {
	V3 random = V3(hiprand_uniform(localDevState), hiprand_uniform(localDevState), hiprand_uniform(localDevState)).normalize().mul_val(2.0).sub(V3(1.0));
	//while (random.squared_length() >= 1) {
	//	random = V3(hiprand_uniform(localDevState), hiprand_uniform(localDevState), hiprand_uniform(localDevState)).mul_val(2.0).sub(V3(1));
	//}
	return random;
}

__device__ V3 PBRMaterial::random_direction_in_n(float radius, hiprandState* localDevState) {
	V3 random = random_direction(localDevState);
	return random.mul_val(radius);
}

// Perfect reflection
__device__ V3 PBRMaterial::reflect(Ray& in_dir, hitReg& hR) {
	V3 normal_dir = in_dir.direction.normalize();
	V3 reflected = normal_dir.sub(hR.normal_vector.mul_val(2 * normal_dir.dot(hR.normal_vector)));
	return reflected;
}

// Imperfect scattering
__device__ V3 PBRMaterial::random_scatter(Ray& in_dir, hitReg& hR, hiprandState* localDevState) {
	return in_dir.get_at(hR.time).add(hR.normal_vector.add(random_direction(localDevState)));
}

__device__ V3 PBRMaterial::hitColor(Ray& in_ray, hitReg& hR, Ray& out_ray, hiprandState* localDevState){
	if (roughness <= 0.0f) {
		out_ray = Ray(in_ray.get_at(hR.time), this->reflect(in_ray, hR));
	}
	else if (roughness >= 1.0f) {
		out_ray = Ray(in_ray.get_at(hR.time), this->random_scatter(in_ray, hR, localDevState).sub(in_ray.get_at(hR.time)));
	}
	else {
		out_ray = Ray(in_ray.get_at(hR.time), this->reflect(in_ray, hR).mul_val((1.0f - this->roughness)).add(this->random_scatter(in_ray, hR, localDevState).mul_val(roughness)));
	}
	return this->base_color.div_val(255.0f);
}