#include "Tracer.cuh"
#include <iostream>
using namespace std;

__device__ V3 Tracer::get_light_intensity(Ray in, Ray secondary, Sphere** objects, V3 hitcolor, AreaLight a, hitReg primHit, PBRMaterial* mat, int numObjects) {
	bool hit_anything = false;
	float closest_so_far = FLT_MAX;
	for (int k = 0; k < numObjects; k++) {
		Sphere current = *(*(objects + k));
		hitReg temp_rec = in.intersect(current.origin, 0.00001f, closest_so_far, current.radius);
		if (temp_rec.hit) {
			hit_anything = true;
			closest_so_far = temp_rec.time;
		}
	}
	if (!hit_anything) {
		hitReg temp_rec = secondary.intersect(a.pos.origin, 0.00001f, closest_so_far, a.pos.radius);
		if (temp_rec.hit) {
			return a.color;
		}
		else {
			// object albedo * light color * dot product between hit normal and light vector
			return hitcolor.mul(((a.color).mul_val((primHit.normal_vector.dot(in.direction.normalize())))).mul_val(mat->roughness));
		}
	}
	else {
		return V3(0, 0, 0);
	}
}

__device__ V3 Tracer::trace_ray(const Ray& ray, Sphere** objects, AreaLight** lights, int max_bounces, int numObjects, int numLights, hiprandState* localDevState) {
	Ray cur_r = ray;
	V3 cur_attenuation = V3(0.0, 0.0, 0.0);
	PBRMaterial* prevMat = nullptr;
	for (int i = 0; i < max_bounces; i++) {
		hitReg hit{ false, 0, V3(0, 0, 0) };
		hitReg lightHit{ false, 0, V3(0, 0, 0) };
		bool hit_anything = false;
		float closest_so_far = FLT_MAX;
		PBRMaterial* current_mat = nullptr;
		for (int j = 0; j < numObjects; j++) {
			Sphere current = *(*(objects + j));
			hitReg temp_rec = cur_r.intersect(current.origin, 0.00001f, closest_so_far, current.radius);
			if (temp_rec.hit) {
				current_mat = current.mat;
				hit_anything = true;
				closest_so_far = temp_rec.time;
				hit = temp_rec;
				hit.hitPoint = cur_r.get_at(hit.time);
			}
		}

		if (hit_anything) {
			Ray secondaryRay = Ray(V3(0, 0, 0), V3(0, 0, 0));
			V3 attenuation = current_mat->hitColor(cur_r, hit, secondaryRay, localDevState);
			V3 true_light_intensity = V3(0.0, 0.0, 0.0);
			if (current_mat->roughness != 0) {
				Ray secondaryRay(V3(0, 0, 0), V3(0, 0, 0));
				V3 attenuation = current_mat->hitColor(cur_r, hit, secondaryRay, localDevState);
				V3 true_color(0, 0, 0);
				for (int j = 0; j < numLights; j++) {
					AreaLight l = *(*(lights + j));
					Ray shadowRay = Ray(hit.hitPoint, hit.hitPoint.add(l.pos.origin.add(current_mat->random_direction_in_n(l.pos.radius, localDevState))));
					true_light_intensity = true_light_intensity.add(get_light_intensity(shadowRay, secondaryRay, objects, attenuation, l, hit, current_mat, numObjects));
					true_light_intensity = true_light_intensity.mul_val(l.get_intensity((hit.hitPoint.distance_to(l.pos.origin))));
				}
				if (i == 0) {
					cur_attenuation = true_light_intensity;
				}
				else {
					cur_attenuation = cur_attenuation.add((true_light_intensity).mul_val(1.0 - prevMat->roughness));

				}
			}
			prevMat = current_mat;
			cur_r = secondaryRay;
		} else {
			//Multiply for global illumination (background color)
			return cur_attenuation.mul_val(255.0);
		}

		//if (hit_anything) {
		//	V3 target = hit.hitPoint.add(hit.normal_vector).add(current_mat->random_direction(localDevState));
		//	V3 att = cur_attenuation;
		//	cur_attenuation = att.mul_val(0.5f);
		//	cur_r = Ray(hit.hitPoint, target.sub(hit.hitPoint));
		//} else {
		//	V3 unit_direction = cur_r.direction.normalize();
		//	float t = 0.5f * (unit_direction.y + 1.0f);
		//	V3 c = V3(1.0, 1.0, 1.0).mul_val(1.0 - t).add(V3(0.5, 0.7, 1.0).mul_val(t));
		//	return c.mul(cur_attenuation).mul_val(255.0);
		//}

		//if (hit_anything) {
		//	return current_mat->base_color;
		//}
		//else {
		//	return V3(0.0, 0.0, 0.0);
		//}

	}
	return V3(0.0, 0.0, 0.0);
}

//__device__ V3 Tracer::trace_ray_2(const Ray& ray, Sphere** objects, AreaLight** lights, int max_bounces, int numObjects, int numLights, hiprandState* localDevState) {
//	Ray cur_r = ray;
//	V3 cur_attenuation = V3(0.0, 0.0, 0.0);
//	PBRMaterial* prevMat = nullptr;
//	for (int i = 0; i < max_bounces; i++) {
//		hitReg hit{ false, 0, V3(0, 0, 0) };
//		hitReg lightHit{ false, 0, V3(0, 0, 0) };
//		bool hit_anything = false;
//		float closest_so_far = FLT_MAX;
//		PBRMaterial* current_mat = nullptr;
//		for (int j = 0; j < 1; j++) {
//			Sphere current = *(*(objects + j));
//			hitReg temp_rec = cur_r.intersect(current.origin, 0.00001f, closest_so_far, current.radius);
//			if (temp_rec.hit) {
//				current_mat = current.mat;
//				hit_anything = true;
//				closest_so_far = temp_rec.time;
//				hit = temp_rec;
//				hit.hitPoint = cur_r.get_at(hit.time);
//			}
//		}
//
//		if (hit_anything) {
//			Ray secondaryRay = Ray(V3(0, 0, 0), V3(0, 0, 0));
//			V3 attenuation = current_mat->hitColor(cur_r, hit, secondaryRay, localDevState);
//			V3 true_light_intensity = V3(0.0, 0.0, 0.0);
//			if (current_mat->roughness != 0) {
//				Ray secondaryRay(V3(0, 0, 0), V3(0, 0, 0));
//				V3 attenuation = current_mat->hitColor(cur_r, hit, secondaryRay, localDevState);
//				V3 true_color(0, 0, 0);
//				for (int j = 0; j < numLights; j++) {
//					AreaLight l = *(*(lights + j));
//					Ray shadowRay = Ray(hit.hitPoint, hit.hitPoint.add(l.pos.origin.add(current_mat->random_direction_in_n(l.pos.radius, localDevState))));
//					true_light_intensity = true_light_intensity.add(get_light_intensity(shadowRay, secondaryRay, objects, attenuation, l, hit, current_mat, numObjects));
//					true_light_intensity = true_light_intensity.mul_val(l.get_intensity((hit.hitPoint.distance_to(l.pos.origin))));
//				}
//				if (i == 0) {
//					cur_attenuation = true_light_intensity;
//				}
//				else {
//					cur_attenuation = cur_attenuation.add((true_light_intensity).mul_val(1.0 - prevMat->roughness));
//
//				}
//			}
//			prevMat = current_mat;
//			cur_r = secondaryRay;
//		}
//		else {
//			//Multiply for global illumination (background color)
//			return cur_attenuation.mul_val(255.0);
//		}
//
//		//if (hit_anything) {
//		//	V3 target = hit.hitPoint.add(hit.normal_vector).add(current_mat->random_direction(localDevState));
//		//	V3 att = cur_attenuation;
//		//	cur_attenuation = att.mul_val(0.5f);
//		//	cur_r = Ray(hit.hitPoint, target.sub(hit.hitPoint));
//		//} else {
//		//	V3 unit_direction = cur_r.direction.normalize();
//		//	float t = 0.5f * (unit_direction.y + 1.0f);
//		//	V3 c = V3(1.0, 1.0, 1.0).mul_val(1.0 - t).add(V3(0.5, 0.7, 1.0).mul_val(t));
//		//	return c.mul(cur_attenuation).mul_val(255.0);
//		//}
//
//		//if (hit_anything) {
//		//	return current_mat->base_color;
//		//}
//		//else {
//		//	return V3(0.0, 0.0, 0.0);
//		//}
//
//	}
//	return V3(0.0, 0.0, 0.0);
//}