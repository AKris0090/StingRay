#include "Tracer.cuh"
#include <execution>
#include <iostream>
using namespace std;

__device__ V3 Tracer::get_light_intensity(Ray in, Sphere** objects, V3 hitcolor, AreaLight a, hitReg primHit, PBRMaterial* mat, int numObjects) {
	bool hit_anything = false;
	float closest_so_far = FLT_MAX;
	for (int k = 0; k < numObjects; k++) {
		Sphere current = *(*(objects + k));
		hitReg temp_rec = in.intersect(current.origin, 0.00001f, closest_so_far, current.radius);
		if (temp_rec.hit) {
			hit_anything = true;
			closest_so_far = temp_rec.time;
		}
	}
	if (!hit_anything) {
		return hitcolor.mul(a.color).mul_val(primHit.normal_vector.dot(in.direction.normalize()));
	}
	else {
		return V3(0, 0, 0);
	}
}

__device__ V3 Tracer::trace_ray(const Ray& ray, Sphere** objects, AreaLight** lights, int max_bounces, int numObjects, int numLights, hiprandState* localDevState) {
	Ray cur_r = ray;
	V3 cur_attenuation = V3(1.0, 1.0, 1.0);
	for (int i = 0; i < max_bounces; i++) {
		hitReg hit{ false, 0, V3(0, 0, 0) };
		hitReg lightHit{ false, 0, V3(0, 0, 0) };
		bool hit_anything = false;
		float closest_so_far = FLT_MAX;
		PBRMaterial* current_mat = nullptr;
		for (int j = 0; j < numObjects; j++) {
			Sphere current = *(*(objects + j));
			hitReg temp_rec = cur_r.intersect(current.origin, 0.00001f, closest_so_far, current.radius);
			if (temp_rec.hit) {
				current_mat = current.mat;
				hit_anything = true;
				closest_so_far = temp_rec.time;
				hit = temp_rec;
				hit.hitPoint = cur_r.get_at(hit.time);
			}
		}

		// SOMETHING WRONG WITH THE REFLECTIONS
		if (hit_anything) {
			V3 true_color(0, 0, 0);
			Ray secondaryRay = Ray(V3(0, 0, 0), V3(0, 0, 0));
			V3 attenuation = current_mat->hitColor(cur_r, hit, secondaryRay, localDevState);
			V3 emittance = attenuation.mul(current_mat->emission_strength);
			for (int j = 0; j < numLights; j++) {
				AreaLight l = *(*(lights + j));
				float intensity = l.get_intensity(hit.hitPoint.distance_to(l.pos.origin));
				Ray shadowRay(cur_r.get_at(hit.time), l.pos.origin.sub(hit.hitPoint).add(current_mat->random_direction_in_n(l.pos.radius, localDevState)));
				true_color = true_color.add(get_light_intensity(shadowRay, objects, attenuation, l, hit, current_mat, numObjects).mul_val(intensity));
				if (secondaryRay.intersect(l.pos.origin, 0.0f, FLT_MAX, l.pos.radius).hit) {
					return l.color.mul(intensity);
				}
			}
			cur_attenuation = attenuation.mul_val(1.0f - current_mat->roughness).mul(cur_attenuation).add(true_color.mul_val(current_mat->roughness).add(emittance));
			cur_r = secondaryRay;
		} else {
			return cur_attenuation;
		}

		//if (hit_anything) {
		//	V3 target = hit.hitPoint.add(hit.normal_vector).add(current_mat->random_direction(localDevState));
		//	V3 att = cur_attenuation;
		//	cur_attenuation = att.mul_val(0.5f);
		//	cur_r = Ray(hit.hitPoint, target.sub(hit.hitPoint));
		//} else {
		//	V3 unit_direction = cur_r.direction.normalize();
		//	float t = 0.5f * (unit_direction.y + 1.0f);
		//	V3 c = V3(1.0, 1.0, 1.0).mul_val(1.0 - t).add(V3(0.5, 0.7, 1.0).mul_val(t));
		//	return c.mul(cur_attenuation).mul_val(255.0);
		//}

		//if (hit_anything) {
		//	return current_mat->base_color;
		//}
		//else {
		//	return V3(0.0, 0.0, 0.0);
		//}

	}
	return V3(0.0, 0.0, 0.0);
}