#include "hip/hip_runtime.h"
#include "Display.cuh"
#include "SDL.h"
#include ""
#include <random>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define SCREEN_WIDTH 1200
#define SCREEN_HEIGHT 600
#define NUMBOUNCES 3

using namespace std;

using std::chrono::high_resolution_clock;
using std::chrono::duration_cast;
using std::chrono::duration;
using std::chrono::milliseconds;

// GPU Error Checking MACRO
#define gpuChk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
        fprintf(stdout, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

// Clamping the color traced
__device__ float clampRGB(float in) {
    if (in < 0.0f) {
        return 0.0f;
    }
    else if (in > 255.0f) {
        return 255.0f;
    }
    else {
        return in;
    }
}

// Setup CUDA resources in device memory. Only once.
__global__ void setup_kernel(Sphere** objects, AreaLight** lights, PBRMaterial** mats, int numLights, int numObjects) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        V3 center_one = { 0, 0, -1 };
        V3 radius_one = 1.5;

        PBRMaterial* metal = new PBRMaterial(V3(204.0f, 204.0f, 204.0f), 0.0f, 0.05f, 0.0f, 0.0f, 0.0f);
        PBRMaterial* metal2 = new PBRMaterial(V3(204.0f, 204.0f, 204.0f), 0.0f, 0.25f, 0.0f, 0.0f, 0.0f);
        PBRMaterial* red = new PBRMaterial(V3(255.0f, 0.0f, 0.0f), 0.0f, 1.0f, 0.0f, 0.0f, 0.0f);
        PBRMaterial* blue = new PBRMaterial(V3(0.0f, 0.0f, 255.0f), 0.0f, 1.0f, 0.0f, 0.0f, 0.0f);

        *(mats) = metal;
        *(mats + 1) = metal2;
        *(mats + 2) = red;
        *(mats + 3) = blue;

        *(lights) = new AreaLight(Sphere(V3(-2.0f, 2.0f, 1.0f), 0.15f), 1500.0f);
        *(lights + 1) = new AreaLight(Sphere(V3(2.0f, 2.0f, 1.0f), 0.15f), 1500.0f);

        *(objects) = new Sphere(V3(0, 0, -1), 0.5, mats[3]);
        *(objects + 1) = new Sphere(V3(0, -100.5, -1), 100, mats[2]);
        *(objects + 2) = new Sphere(V3(1, 0, -1), 0.5, mats[0]);
        *(objects + 3) = new Sphere(V3(-1, 0, -1), 0.5, mats[1]);
    }
}


__global__ void updateDisplay(V3* totals, V3* devPixels, V3 hor, V3 ver, V3 botL, V3 copOrigin, float numSamples, const int numBounces, int numObjects, Sphere** objects, int numLights, AreaLight** lights, hiprandState* devStates, int repeatSamples, unsigned long seed) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    int index = i + (j * SCREEN_WIDTH);
    hiprand_init(seed, index, 0, &devStates[index]);
    hiprandState localDevState = devStates[index];
    if ((i >= SCREEN_WIDTH) || (j >= SCREEN_HEIGHT)) return;

    V3 u = hor.mul_val((float)(i + (hiprand_uniform(&localDevState))) / (float) SCREEN_WIDTH);
    V3 v = ver.mul_val((float)(j + (hiprand_uniform(&localDevState))) / (float) SCREEN_HEIGHT);

    Ray primary_ray(copOrigin, botL.add(u).add(v).sub(copOrigin));
    V3 ret_color = Tracer::trace_ray(primary_ray, objects, lights, numBounces, numObjects, numLights, &localDevState);

    totals[index].x += clampRGB(ret_color.x);
    totals[index].y += clampRGB(ret_color.y);
    totals[index].z += clampRGB(ret_color.z);
    devPixels[index] = totals[index].div_val(repeatSamples);
}

int main(int argc, char** arcgv) {
    int tx = 8;
    int ty = 8;
    dim3 blocks(SCREEN_WIDTH / tx + 1, SCREEN_HEIGHT / ty + 1);
    dim3 threads(tx, ty);

    DisplayWindow window;

    int numObjects, numLights;
    bool running = true;
    SDL_Event event;
    float numSamples = 1000.0f;

    window.initDisplay(SCREEN_WIDTH, SCREEN_HEIGHT);

    numObjects = 4;
    numLights = 2;

    hiprandState* devStates;
    gpuChk(hipMallocManaged((void**)&devStates, (SCREEN_WIDTH * SCREEN_HEIGHT) * sizeof(hiprandState)));

    gpuChk(hipMallocManaged((void**)&(window.objects), numObjects * sizeof(Sphere*)));
    gpuChk(hipMallocManaged((void**)&(window.lights), numLights * sizeof(AreaLight*)));
    gpuChk(hipMallocManaged((void**)&(window.mats), 4 * sizeof(PBRMaterial*)));
    gpuChk(hipMallocManaged((void**)&(window.totals), ((SCREEN_WIDTH * SCREEN_HEIGHT) * sizeof(V3))));
    gpuChk(hipMallocManaged((void**)&(window.devPixels), ((SCREEN_WIDTH * SCREEN_HEIGHT) * sizeof(V3))));


    float cam_aspect_width = 4;
    float cam_aspect_height = 2;
    V3* copyTotals;

    window.bot_left = V3(-cam_aspect_width, -cam_aspect_height, -1);
    window.horizontal = V3(cam_aspect_width * 2, 0, 0);
    window.vertical = V3(0, cam_aspect_height * 2, 0);

    // setup seeds
    setup_kernel<<<1, 1>>>(window.objects, window.lights, window.mats, numLights, numObjects);
    gpuChk(hipDeviceSynchronize());
    gpuChk(hipPeekAtLastError());

    window.copied_origin = V3(0.0, 0.0, 0.0);

    while (SDL_PollEvent(&event) || running) {

        cout << "\r";

        auto t1 = high_resolution_clock::now();


        Uint32* pixels = (Uint32*) malloc((SCREEN_HEIGHT * SCREEN_WIDTH) * sizeof(Uint32));
        window.texture = SDL_CreateTexture(window.renderer, SDL_PIXELFORMAT_ARGB8888, SDL_TEXTUREACCESS_STREAMING, SCREEN_WIDTH, SCREEN_HEIGHT);

        if (window.repeat_samples < numSamples) {

            window.repeat_samples += 1;
            updateDisplay<<<blocks, threads>>>(window.totals, window.devPixels, window.horizontal, window.vertical, window.bot_left, window.copied_origin, numSamples, NUMBOUNCES, numObjects, window.objects, numLights, window.lights, devStates, window.repeat_samples, unsigned(rand()));
            gpuChk(hipDeviceSynchronize());
            gpuChk(hipPeekAtLastError());

            copyTotals = (V3*)malloc((SCREEN_WIDTH * SCREEN_HEIGHT) * sizeof(V3));

            gpuChk(hipMemcpy(copyTotals, window.devPixels, ((SCREEN_WIDTH * SCREEN_HEIGHT) * sizeof(V3)), hipMemcpyDeviceToHost));

            for (int i = 0; i < SCREEN_WIDTH; i++) {
                for (int j = 0; j < SCREEN_HEIGHT; j++) {
                    int index = i + (j * SCREEN_WIDTH);
                    pixels[index] = SDL_MapRGB(window.surface->format, (Uint8)(copyTotals[index].x), (Uint8)(copyTotals[index].y), (Uint8)(copyTotals[index].z));
                }
            }

            free(copyTotals);
            SDL_UpdateTexture(window.texture, NULL, pixels, SCREEN_WIDTH * sizeof(Uint32));
            SDL_RenderClear(window.renderer);
            SDL_RenderCopyEx(window.renderer, window.texture, NULL, NULL, 0, NULL, SDL_FLIP_VERTICAL);
            SDL_DestroyTexture(window.texture);
            SDL_RenderPresent(window.renderer);
        }
        auto t2 = high_resolution_clock::now();

        /* Getting number of milliseconds as an integer. */
        auto ms_int = duration_cast<milliseconds>(t2 - t1);

        /* Getting number of milliseconds as a double. */
        duration<double, std::milli> ms_double = t2 - t1;

        cout << window.repeat_samples << "/" << numSamples << " samples " << ms_double.count() << " ms";

        free(pixels);

        switch (event.type) {
        case SDL_QUIT:
            running = false;
            break;
        default:
            break;
        }
    }
    SDL_DestroyRenderer(window.renderer);
    SDL_DestroyWindow(window.window);
    hipFree(window.totals);
    hipFree(devStates);

    hipFree(window.lights);
    hipFree(window.objects);
    hipFree(window.mats);

    return 0;
}