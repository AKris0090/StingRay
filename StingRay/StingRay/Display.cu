﻿#include "Display.cuh"

#define SDL_MAIN_HANDLED

void DisplayWindow::initDisplay(int screen_width, int screen_height) {
    cout << ",d88~~\\ ~~~888~~~ 888 888b    |  e88~~\\  888~-_        e      Y88b    / " << endl;
    cout << "8888       888    888 |Y88b   | d888     888   \\      d8b      Y88b  /  " << endl;
    cout << "`Y88b      888    888 | Y88b  | 8888 __  888    |    /Y88b      Y88b/  " << endl;
    cout << " `Y88b,    888    888 |  Y88b | 8888   | 888   /    /  Y88b      Y8Y  " << endl;
    cout << "   8888    888    888 |   Y88b| Y888   | 888_-~    /____Y88b      Y   " << endl;
    cout << "\\__88P'    888    888 |    Y888  \"88__ / 888 ~- _ /      Y88b    /    " << endl;
    cout << "v 1.0.0 -------------------------------------------------------------   " << endl;
    cout << "--------------------------------------------------------------------- " << endl;

    // Startup the video feed
    SDL_Init(SDL_INIT_VIDEO);

    // Create the SDL Window and open
    window = SDL_CreateWindow("StingRay", SDL_WINDOWPOS_UNDEFINED, SDL_WINDOWPOS_UNDEFINED, (screen_width), (screen_height), 0);

    this->SCREEN_HEIGHT = screen_height;
    this->SCREEN_WIDTH = screen_width;
        
    // Create the renderer for the window
    renderer = SDL_CreateRenderer(window, -1, SDL_RENDERER_ACCELERATED);

    // Get surface off of the window
    surface = SDL_GetWindowSurface(window);
}