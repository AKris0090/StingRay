#include "hip/hip_runtime.h"
#include "Ray.cuh"
#include <iostream>

__device__ hitReg Ray::intersect(V3 center, float min_t, float max_t, float radius) {
	hit = { false, 0.0f, V3(0.0f, 0.0f, 0.0f), V3(0.0f, 0.0f, 0.0f) };
	oc = this->origin.sub(center);
	a = this->direction.dot(this->direction);
	b = oc.dot(this->direction);
	c = oc.dot(oc) - (radius * radius);
	disc = b * b - a * c;
	if (disc > 0) {
		temp_quad_f = ((0 - b) - sqrt(b * b - a * c)) / a;
		if (temp_quad_f < max_t && temp_quad_f > min_t) {
			hit.time = temp_quad_f;
			hit.hit = true;
			hit.normal_vector = this->get_at(temp_quad_f).sub(center).div_val(radius);
			return hit;
		}
		temp_quad_f = ((0 - b) + sqrt(b * b - a * c)) / a;
		if (temp_quad_f < max_t && temp_quad_f > min_t) {
			hit.time = temp_quad_f;
			hit.hit = true;
			hit.normal_vector = this->get_at(temp_quad_f).sub(center).div_val(radius);
			return hit;
		}
	}
	return hit;
}

//__device__ hitReg Ray::intersect(V3 v1, V3 v2, V3 v3, V3 normal, float min_t, float max_t) {
//	hitReg h;
//
//	float ndotDir = normal.dot(this->direction);
//	if (abs(ndotDir) < 0.000000001f) {
//		return h;
//	}
//
//	float d = normal.mul_val(-1.0).dot(v1);
//
//	h.time = -(normal.dot(this->origin) + d) / ndotDir;
//
//	if (t < 0) {
//		return h;
//	}
//
//	h.hitPoint = this->origin.add(this->direction.mul_val(t));
//
//	V3 C;
//
//	V3 edge1 = v2.sub(v1);
//	V3 vp0 = h.hitPoint.sub(v1);
//	C = edge1.cross(vp0);
//	if (normal.dot(C) < 0) {
//		return h;
//	}
//
//	V3 edge2 = v2.sub(v2);
//	V3 vp1 = h.hitPoint.sub(v2);
//	C = edge2.cross(vp1);
//	if (normal.dot(C) < 0) {
//		return h;
//	}
//
//	V3 edge3 = v1.sub(v3);
//	V3 vp2 = h.hitPoint.sub(v3);
//	C = edge3.cross(vp2);
//	if (normal.dot(C) < 0) {
//		return h;
//	}
//
//	h.hit = true;
//	h.normal_vector = normal;
//	
//	return h;
//}

__device__ Ray Ray::copy() {
	return Ray(this->origin, this->direction);
}