#include "hip/hip_runtime.h"
#include "Ray.cuh"
#include <iostream>

__device__ hitReg Ray::intersect(V3 center, float min_t, float max_t, float radius) {
	hit = { false, 0.0f, V3(0.0f, 0.0f, 0.0f), V3(0.0f, 0.0f, 0.0f) };
	oc = this->origin.sub(center);
	a = this->direction.dot(this->direction);
	b = oc.dot(this->direction);
	c = oc.dot(oc) - (radius * radius);
	disc = b * b - a * c;
	if (disc > 0) {
		temp_quad_f = ((0 - b) - sqrt(b * b - a * c)) / a;
		if (temp_quad_f < max_t && temp_quad_f > min_t) {
			hit.time = temp_quad_f;
			hit.hit = true;
			hit.normal_vector = this->get_at(temp_quad_f).sub(center).div_val(radius);
			return hit;
		}
		temp_quad_f = ((0 - b) + sqrt(b * b - a * c)) / a;
		if (temp_quad_f < max_t && temp_quad_f > min_t) {
			hit.time = temp_quad_f;
			hit.hit = true;
			hit.normal_vector = this->get_at(temp_quad_f).sub(center).div_val(radius);
			return hit;
		}
	}
	return hit;
}

__device__ Ray Ray::copy() {
	return Ray(this->origin, this->direction);
}